#include<hip/hip_runtime.h>

#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <cinttypes>


const double pi = std::atan(1) / 4;
const double delta = 1e-4;
const double eps = 1e-10;

double Ax = -0.353, Bx = 0.353, Ay = 0.3, By = Ay, C = 3 * pi / 8;

struct VarsToShare {
  double Ax;
  double Bx;
  double Ay;
  double By;
  double C;
  double Pi;
  double Delta;
  double Eps;
} typedef Vars;

__device__ double calculateDistance(double* x0, double* x1, size_t n) {
  double sum = 0;
  for (size_t i = 0; i < n; ++i) {
    sum += std::pow(x0[i] - x1[i], 2);
  }
  return std::sqrt(sum);
}

__device__ void calculateF(double* result, double* input, Vars* vars, bool isSequential) {
   if (isSequential) {
    // printf("Sequential = true\n");
    result[0] = input[0] + input[2] * std::cos(1.5 * vars->Pi - input[3]) - vars->Ax;
    result[1] = input[1] + input[2] * std::cos(1.5 * vars->Pi + input[4]) - vars->Bx;
    result[2] = input[2] + input[2] * std::sin(1.5 * vars->Pi - input[3]) - vars->Ay;
    result[3] = (input[3] + input[4]) * input[2] + (input[1] - input[0]) - vars->C;
    result[4] = input[2] + input[2] * std::sin(1.5 * vars->Pi + input[4]) - vars->By;
  } else {
    std::uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadIdx.x == 0) {
      // printf("Thread 0\n");
      // printf("Before:Res_0 = %.14lf\n", result[0]);
      result[0] = input[0] + input[2] * std::cos(1.5 * vars->Pi - input[3]) - vars->Ax;
      // printf("After:Res_0 = %.14lf\n", result[0]);
    } else if (threadIdx.x == 1){
      // printf("Thread 1\n");
      // printf("Before:Res_1 = %.14lf\n", result[1]);
      // printf("input[1] = %lf input[2] = %lf std::cos(1.5 * vars->Pi + input[4]) = %lf vars->Bx = %lf\n",input[1], input[2], std::cos(1.5 * vars->Pi + input[4]), vars->Bx);

      result[1] = input[1] + input[2] * std::cos(1.5 * vars->Pi + input[4]) - vars->Bx;
      // printf("After:Res_1 = %.14lf\n", result[1]);
    } else if (threadIdx.x == 2) {
      // printf("Thread 2\n");
      // printf("Before:Res_2 = %.14lf\n", result[2]);
      result[2] = input[2] + input[2] * std::sin(1.5 * vars->Pi - input[3]) - vars->Ay;
      // printf("After:Res_2 = %.14lf\n", result[2]);
    } else if (threadIdx.x == 3) {
      // printf("Thread 3\n");
      // printf("Before:Res_3 = %.14lf\n", result[3]);
      result[3] = (input[3] + input[4]) * input[2] + (input[1] - input[0]) - vars->C;
      // printf("After:Res_3 = %.14lf\n", result[3]);
    } else if (threadIdx.x == 4) {
      // printf("Thread 4\n");
      // printf("Before:Res_4 = %.14lf\n", result[4]);
      result[4] = input[2] + input[2] * std::sin(1.5 * vars->Pi + input[4]) - vars->By;
      // printf("After:Res_4 = %.14lf\n", result[4]);
    }
    __syncthreads();
  }
}

__device__ void print_progress(unsigned step, double* x0, double* x1) {
  printf("-------------------\n");
  printf("Progress\n");
  printf("Step: %u\n", step);
  printf("x1: %lf %lf\n", x0[0], x1[0]);
  printf("x2: %lf %lf\n", x0[1], x1[1]);
  printf("y: %lf %lf\n", x0[2], x1[2]);
  printf("phi1: %lf %lf\n", x0[3], x1[3]);
  printf("phi2: %lf %lf\n", x0[4], x1[4]);
  printf("-------------------\n");
}

__global__ void calculateValue(double* x0, double* x1, Vars* vars, size_t n, bool isSequential) {
  __shared__ unsigned count;
  double* FValue = new double[n];
  while(true) {
    calculateF(FValue, x0, vars, isSequential);
    __barrier_sync(0);
    if (threadIdx.x == 0) {
      for (size_t i = 0; i < n; ++i) {
        printf("FValue[%lu] %lf\n", i, FValue[i]);
        x1[i] = x0[i] - FValue[i] * vars->Delta;
      }
    }
    __barrier_sync(0);
    if (threadIdx.x == 0) atomicAdd(&count, 1);
    __barrier_sync(0);
    if (!(count%5000) && (threadIdx.x == 0)) print_progress(count, x0, x1);
    __barrier_sync(0);
    if (calculateDistance(x0, x1, n) < vars->Eps) break;
    if (threadIdx.x == 0) {
      for (size_t i = 0; i < n; ++i) {
        x0[i] = x1[i];
      }
    }
    __barrier_sync(0);
  }
  delete[] FValue;
}

__host__ void print_result(double* x) {
    printf("-------------------\n");
    printf("Result\n");
    printf("x1 : %lf\n", x[0]);
    printf("x2 : %lf\n", x[1]);
    printf("y : %lf\n", x[2]);
    printf("phi1 : %lf\n", x[3]);
    printf("phi2 : %lf\n", x[4]);
    printf("-------------------\n");
    printf("F(x) = {%.10e, %.10e, %.10e, %.10e, %.10e}\n",
      x[0] + x[2] * std::cos(1.5 * pi - x[3]) - Ax,
      x[1] + x[2] * std::cos(1.5 * pi + x[4]) - Bx,
      x[2] + x[2] * std::sin(1.5 * pi - x[3]) - Ay,
      (x[3] + x[4]) * x[2] + (x[1] - x[0]) - C,
      x[2] + x[2] * std::sin(1.5 * pi + x[4]) - By
    );
}

int main() {
  Vars* vars;
  const int NumOfEquations = 5;
  double *x0, *x1;
  hipMallocManaged(&x0, sizeof(double) * NumOfEquations);
  hipMallocManaged(&x1, sizeof(double) * NumOfEquations);
  //preassign vals
  x0[0] = -0.1;
  x0[1] = 0.1;
  x0[2] = 0.0;
  x0[3] = 2.0;
  x0[4] = 2.0;

  x1[0] = 0.0;
  x1[1] = 0.0;
  x1[2] = 0.0;
  x1[3] = 0.0;
  x1[4] = 0.0;
  hipMallocManaged(&vars, sizeof(Vars));
  vars->Ax = Ax, vars->Ay = Ay, 
  vars->Bx = Bx, vars->By = By, 
  vars->C = C, vars->Pi = pi, 
  vars->Delta = delta, vars->Eps = eps;

  int numBlocks = 1, numThreadsPerBlock = 5;
  bool isSequential = ((numBlocks == 1) && (numThreadsPerBlock == 1)) ? true : false;

  calculateValue<<<numBlocks, numThreadsPerBlock>>>(x0, x1, vars, NumOfEquations, isSequential);
  hipDeviceSynchronize();
  print_result(x0);
  hipFree(&x0);
  hipFree(&x1);
  return 0;
}